/*
 *  test_rss.cu
    @description : runs tests on the RSS data structure by using Rectangle and
 Triangles tests

    @author Atulya Shivam Shree
    Created on: Dec 12, 2017
    Copyright (c) 2017 Atulya Shivam Shree
 */

#include "../compile_CUDA.h"

#include <cassert>
#include <iomanip>
#include <iostream>
#include <algorithm>
#include <vector>

using std::cout;
using std::endl;
using std::string;
using std::vector;

#include "../Triangle.h"
#include "../RSS.h"
#include "../BVH.h"

__host__ float distTrianglesGPU(const Triangle *, const Triangle *, 
                  const float R[3][3], const float t[3],
                  DistTriangleVars* vars);

__host__ float distRSSGPU(const float R[3][3], const float t[3],
                            const RSS* r1, const RSS* r2,
                                DistRSSVars* vars);

#define DIST_TRIANGLES distTrianglesGPU
// use the test cases on the GPU triangles function
#define DIST_RSS distRSSGPU

#include "../Rectangle_tests.h"
#include "../Triangle-cuda-inl.h"
#include "../RSS-cuda-inl.h"

const int size_rss = sizeof(RSS);

// single case timing
__host__ void testSingleTiming();
// multiple instances of same in sequence
__host__ void testMultipleSerial();
// multiple instances of same in an array
__host__ void testMultipleSame();
// Check collision within an array of random triangles
__host__ void testMultipleRandom();

__host__ int main(int argc, char *argv[]) {
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  cout << "Device name: " << prop.name << endl;
  cout << "float size is " << sizeof(float) << endl;

  test_rectangles_2D();
  test_rectangles_3D();
  test_stress_random_RSS();

  print_Stats();

  testSingleTiming();
  // testMultipleSerial();
  // testMultipleSame();
  testMultipleRandom();
}

__host__ float distTrianglesGPU(const Triangle *, const Triangle *, 
                  const float R[3][3], const float t[3],
                  DistTriangleVars* vars)
{
  return 0;
}

// use this function to verify working by running it against the test cases
__host__ float distRSSGPU(const float R[3][3], const float t[3],
                            const RSS* r1, const RSS* r2,
                                DistRSSVars* vars) {
  RSS *d_r1, *d_r2;
  Config *d_cfg;
  Config h_cfg;

  for(int i = 0; i < 3; i++)
  {
    h_cfg.t[i] = t[i];
    for(int j = 0; j < 3; j++)
      h_cfg.R[i][j] = R[i][j];
  }

  hipMalloc(&d_cfg, sizeof(Config));
  hipMalloc(&d_r1, size_rss);
  hipMalloc(&d_r2, size_rss);
  RSSResult *d_res;
  RSSResult h_res;
  hipMalloc(&d_res, sizeof(RSSResult));

  hipMemcpy(d_cfg, &h_cfg, sizeof(Config), hipMemcpyHostToDevice);
  hipMemcpy(d_r1, r1, size_rss, hipMemcpyHostToDevice);
  hipMemcpy(d_r2, r2, size_rss, hipMemcpyHostToDevice);

  computeDistanceSingle<<<1, 1>>>(d_r1, d_r2, d_cfg, d_res);

  hipMemcpy(&h_res, d_res, sizeof(RSSResult), hipMemcpyDeviceToHost);

  hipFree(d_r1);
  hipFree(d_r2);
  hipFree(d_res);

  return h_res.dist;
}

__host__ void testSingleTiming() {
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  RSS h_r1, h_r2;
  RSS *d_r1, *d_r2;

  h_r1 = getRandomRSS();
  h_r2 = getRandomRSS();

  Config *d_cfg;
  Config h_cfg;

  for(int i = 0; i < 3; i++)
  {
    h_cfg.t[i] = t0_[i];
    for(int j = 0; j < 3; j++)
      h_cfg.R[i][j] = matI_[i][j];
  }

  hipMalloc(&d_cfg, sizeof(Config));
  hipMalloc(&d_r1, size_rss);
  hipMalloc(&d_r2, size_rss);
  RSSResult *d_res;
  RSSResult h_res;
  hipMalloc(&d_res, sizeof(RSSResult));

  double t_init = get_wall_time();

  hipMemcpy(d_cfg, &h_cfg, sizeof(Config), hipMemcpyHostToDevice);
  hipMemcpy(d_r1, &h_r1, size_rss, hipMemcpyHostToDevice);
  hipMemcpy(d_r2, &h_r2, size_rss, hipMemcpyHostToDevice);

  computeDistanceSingle<<<1, 1>>>(d_r1, d_r2, d_cfg, d_res);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(&h_res, d_res, sizeof(RSSResult), hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);

  cout << std::setprecision(5);
  // cout << "Time for results using CUDA Event: " << t_results << "ms" << endl;
  cout << "Wall time (Single): " << (t_cuda_end - t_init) * 1000 << "ms"
       << endl;
  cout << endl;

  hipFree(d_r1);
  hipFree(d_r2);
  hipFree(d_res);
}

void testMultipleRandom() {
  srand(static_cast<unsigned>(time(NULL)));
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  RSS *h_r1, *h_r2;
  RSS *d_r1, *d_r2;

  h_r1 = new RSS[NUM_CHECK];
  h_r2 = new RSS[NUM_CHECK];

  if (!h_r1 || !h_r2) {
    cout << "Could not initialize h_r, out of memory" << endl;
    return;
  }

  cout << "Initializing the RSS ... ";
  for (int i = 0; i < NUM_CHECK ; i++) {
    h_r1[i] = getRandomRSS();
    h_r2[i] = getRandomRSS();
  }
  cout << "Done " << endl;
  // cout << "s1 " << h_r1[0];
  // cout << "s2 " << h_r2[0];
  // cout << "dist is " << actual_res << endl;

  // cout << "L: s1 " << h_r1[NUM_CHECK-1];
  // cout << "L: s2 " << h_r2[NUM_CHECK-1];
  // cout << "L: dist is " << distRSSs_fcl(h_r1[NUM_CHECK-1],
  //                            h_r2[NUM_CHECK-1]) << endl;
  Config *d_cfg;
  Config h_cfg;

  for(int i = 0; i < 3; i++)
  {
    h_cfg.t[i] = t1_[i];
    for(int j = 0; j < 3; j++)
      h_cfg.R[i][j] = matR1_[i][j];
  }

  hipMalloc(&d_cfg, sizeof(Config));
  hipMalloc(&d_r1, NUM_CHECK * size_rss);
  hipMalloc(&d_r2, NUM_CHECK * size_rss);
  RSSResult *d_res;
  RSSResult *h_res;
  h_res = new RSSResult[NUM_CHECK];
  hipMalloc(&d_res, NUM_CHECK * sizeof(RSSResult));

  double t_init = get_wall_time();

  hipMemcpy(d_cfg, &h_cfg, sizeof(Config), hipMemcpyHostToDevice);
  hipMemcpy(d_r1, h_r1, NUM_CHECK * size_rss, hipMemcpyHostToDevice);
  hipMemcpy(d_r2, h_r2, NUM_CHECK * size_rss, hipMemcpyHostToDevice);

  int numBlocks = (NUM_CHECK - 1) / BLOCKSIZE_RECT + 1;
  cout << "numBlocks is " << numBlocks << endl;
  dim3 dimBlock(1, BLOCKSIZE_RECT);
  dim3 dimGrid(1, numBlocks);
  computeDistanceArray<<<dimGrid, dimBlock>>>(d_r1, d_r2, d_cfg, d_res,
                                              NUM_CHECK);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(h_res, d_res, NUM_CHECK * sizeof(RSSResult),
             hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);
  cout << std::setprecision(7);
  cout << "Evaluating reslts now ... \n";
  int count_correct = 0;
  for (int i = 0; i < NUM_CHECK; i++) {
    float actual_res = distRectangles_fcl(h_cfg.R, h_cfg.t, h_r1[i], h_r2[i]);

    if (approx_equal(h_res[i].dist, actual_res)) 
      count_correct++;
    else
      cout << i << "] DIFF actual: " << actual_res << ", obtained: " << h_res[i].dist<<endl;
  }

  cout << "total time with copying took " << (get_wall_time() - t_cuda_end) * 1000 << "ms" << endl;

  cout << count_correct << "/" << NUM_CHECK << " are correct" << endl;
  cout << "Wall time multiple random(" << NUM_CHECK
       << "): " << (t_cuda_end - t_init) * 1000 << "ms" << endl;
  cout << endl;

  hipFree(d_r1);
  hipFree(d_r2);
  hipFree(d_res);

  delete[] h_r1;
  delete[] h_r2;
  delete[] h_res;
}
