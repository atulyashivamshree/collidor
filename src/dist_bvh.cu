/*
 *  dist_bvh.cu
    @description : checks triangle intersection on CUDA

    @author Atulya Shivam Shree
    Created on: Dec 12, 2017
    Copyright (c) 2017 Atulya Shivam Shree
 */

#include "compile_CUDA.h"

#include <cassert>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <map>
#include <string>

#include "BVH-cuda-inl.h"
#include "utils/parse_utils.h"
#include "utils/fcl_utility.h"

using std::cout;
using std::endl;
using std::string;
using std::vector;

__host__ void help_message();
__host__ void printResult(std::ostream& os, const vector<DistanceResultGPU> res,
                          const vector<float> elap_time);

__host__ int main(int argc, char* argv[]) {
  if (argc < 2) {
    help_message();
    exit(EXIT_FAILURE);
  }

  // Print device properties
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  cout << "Device name: " << prop.name << endl;

  // Load all configuration params from the file
  Config def_cfg({0.04, 15, {1, 0, 0, 0, 1, 0, 0, 0, 1}, {0, 0, 0}, 1});
  map<string, string> params;
  loadConfig(params, argv[1]);
  def_cfg.gamma = std::stod(params["gamma"]);
  def_cfg.enable_distance_reduction = std::stoi(params["compute_min_dist"]);
  def_cfg.max_iter = std::stoi(params["max_iter"]);
  def_cfg.max_bfs_proc = std::stoi(params["max_bfs_proc"]);
  def_cfg.max_dfs_proc = std::stoi(params["max_dfs_proc"]);

  // Load the two BVH in consideration
  BVH bvhA, bvhB;
  loadOBJToBVH(params["file1"], &bvhA);
  loadOBJToBVH(params["file2"], &bvhB);

  // Load the transformations
  vector<Transform3f> transforms;
  loadTransformations(transforms, params["transforms"]);

  // Compute distances and store the result
  vector<float> elap_time;
  vector<DistanceResultGPU> results = computeDistance(
      &bvhA, &bvhB, def_cfg, transforms, params["outp_prefix"], elap_time);

  // Store the entire result in an output file
  std::ofstream fout;
  string outfile = params["outp_prefix"];
  outfile += ".out";
  fout.open(outfile.c_str());
  printResult(fout, results, elap_time);
  fout.close();

  // Print the final stats of the results
  for (const auto res : results) {
    cout << "==== RESULTS ====" << endl;
    cout << "dist: " << res.dist << " stop: " << res.stop
         << " num_iter: " << res.num_iter << endl;
    cout << "i1: " << res.tsk.i1 << " i2: " << res.tsk.i2
         << " d: " << res.tsk.dist << endl;
    cout << "i1: " << res.tsk2.i1 << " i2: " << res.tsk2.i2
         << " d: " << res.tsk2.dist << endl;
    cout << "idx: " << res.idx << " idy: " << res.idy << endl;
  }

  // Free up memory
  deleteBVH(&bvhA);
  deleteBVH(&bvhB);
}

__host__ void help_message() {
  cout << "Usage : ./dist_bvh.exe FILE.yaml" << endl;
  cout << "FILE.yaml : config file for the run" << endl;
}

__host__ void printResult(std::ostream& os,
                          const vector<DistanceResultGPU> results,
                          const vector<float> elap_time) {
  os << std::setprecision(7);
  for (int i = 0; i < results.size(); i++)
    os << "[" << i << "] " << results[i].dist << " " << elap_time[i] << endl;
}
