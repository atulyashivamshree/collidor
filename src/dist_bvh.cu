/*
 *  dist_bvh.cu
    @description : checks triangle intersection on CUDA

    @author Atulya Shivam Shree
    Created on: Dec 12, 2017
    Copyright (c) 2017 Atulya Shivam Shree
 */

#include "compile_CUDA.h"

#include <cassert>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <map>
#include <string>

#include "BVH-cuda-inl.h"
#include "utils/parse_utils.h"

using std::cout;
using std::endl;
using std::string;
using std::vector;

__host__ void loadBVHData(BVH& bvh, const string filename);
__host__ void help_message();
__host__ void printResult(std::ostream& os, const vector<DistanceResult> res,
                          const vector<float> elap_time);

__host__ int main(int argc, char* argv[]) {
  if (argc < 2) {
    help_message();
    exit(EXIT_FAILURE);
  }

  // Print device properties
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  cout << "Device name: " << prop.name << endl;

  // Load all configuration params from the file
  Config def_cfg({0.04, 15, {1, 0, 0, 0, 1, 0, 0, 0, 1}, {0, 0, 0}, 1});
  map<string, string> params;
  loadConfig(params, argv[1]);
  def_cfg.gamma = std::stod(params["gamma"]);
  def_cfg.enable_distance_reduction = std::stoi(params["compute_min_dist"]);
  def_cfg.max_iter = std::stoi(params["max_iter"]);
  def_cfg.max_bfs_proc = std::stoi(params["max_bfs_proc"]);
  def_cfg.max_dfs_proc = std::stoi(params["max_dfs_proc"]);

  // Load the two BVH in consideration
  BVH bvhA, bvhB;
  loadBVHData(bvhA, params["file1_bvh"]);
  loadBVHData(bvhB, params["file2_bvh"]);

  // Load the transformations
  vector<Transform3f> transforms;
  loadTransformations(transforms, params["transforms"]);

  // Compute distances and store the result
  vector<float> elap_time;
  vector<DistanceResult> results = computeDistance(
      &bvhA, &bvhB, def_cfg, transforms, params["outp_prefix"], elap_time);

  // Store the entire result in an output file
  std::ofstream fout;
  string outfile = params["outp_prefix"];
  outfile += ".out";
  fout.open(outfile.c_str());
  printResult(fout, results, elap_time);
  fout.close();

  // Print the final stats of the results
  for (const auto res : results) {
    cout << "==== RESULTS ====" << endl;
    cout << "dist: " << res.dist << " stop: " << res.stop
         << " num_iter: " << res.num_iter << endl;
    cout << "i1: " << res.tsk.i1 << " i2: " << res.tsk.i2
         << " d: " << res.tsk.dist << endl;
    cout << "i1: " << res.tsk2.i1 << " i2: " << res.tsk2.i2
         << " d: " << res.tsk2.dist << endl;
    cout << "idx: " << res.idx << " idy: " << res.idy << endl;
  }

  // Free up memory
  deleteBVH(&bvhA);
  deleteBVH(&bvhB);
}

__host__ void help_message() {
  cout << "Usage : ./dist_bvh.exe FILE.yaml" << endl;
  cout << "FILE.yaml : config file for the run" << endl;
}

__host__ void printResult(std::ostream& os,
                          const vector<DistanceResult> results,
                          const vector<float> elap_time) {
  os << std::setprecision(7);
  for (int i = 0; i < results.size(); i++)
    os << "[" << i << "] " << results[i].dist << " " << elap_time[i] << endl;
}

HOST_PREFIX void loadBVHData(BVH& bvh, const string filename) {
  std::ifstream fin;
  fin.open(filename.c_str());
  if (!fin.is_open()) {
    cout << "BVH: " << filename << " could not be opened" << endl;
    exit(EXIT_FAILURE);
  }

  loadBVH(fin, &bvh);
}
