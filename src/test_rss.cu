#include "hip/hip_runtime.h"
// @author : Atulya Shivam Shree
// @description : checks triangle intersection on CUDA

#include "compile_CUDA.h"

#include <cassert>
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <time.h>
#include <sys/time.h>

#include "RSS.h"
float distTrianglesGPU(const Triangle*, const Triangle* , DistTriangleVars*);

float distRSSGPU(const Matrix3* R, const Vector3* t,
            const RSS* a, const RSS* b, DistRSSVars* d);

#define DIST_TRIANGLES distTrianglesGPU
// use the test cases on the GPU triangles function
#define DIST_RSS distRSSGPU
#include "Rectangle_tests.h"

using namespace std;

const int size_rss = sizeof(RSS);
const int BLOCKSIZE = 32;

double get_wall_time();
// single case timing
void testSingleTiming();
// multiple instances of same in sequence
void testMultipleSerial();
// multiple instances of same in an array
void testMultipleSame();
// Check collision within an array of random triangles
void testMultipleRandom();

struct Result
{
  float dist;
};

__global__ void computeDistance(const Matrix3* R, const Vector3* t,
                    const RSS *r1, const RSS* d2, Result* res)
{
  __shared__ Matrix3 loc_R;
  __shared__ Vector3 loc_t;
  __shared__ RSS loc_r1;
  __shared__ RSS loc_d2;
  __shared__ DistRSSVars vars;

  loc_R = *R;
  loc_t = *t;

  loc_r1 = *r1;
  loc_d2 = *d2;
  // res->dist = 1e-6 + distRSSs(r1, d2, &vars);
  float dist = rssDistance(&loc_R, &loc_t, &loc_r1, &loc_d2, &vars);
  res->dist = dist;

}

__global__ void computeDistanceArray(const Matrix3* R, const Vector3* t,
                    const RSS *arr_r1, const RSS* arr_d2, 
                      Result* arr_res, int n)
{
  int t_j = threadIdx.y;
  int g_j = blockIdx.y * blockDim.y + threadIdx.y;

  __shared__ Matrix3 loc_R;
  __shared__ Vector3 loc_t;
  __shared__ RSS r1[BLOCKSIZE];
  __shared__ RSS d2[BLOCKSIZE];
  __shared__ DistRSSVars vars[BLOCKSIZE];

  if(threadIdx.y == 0)
  {
    loc_R = *R;
    loc_t = *t;
  }

  if(g_j < n)
  {
    r1[t_j] = arr_r1[g_j];
    d2[t_j] = arr_d2[g_j];
    // res->dist = 1e-6 + distRSSs(r1, d2, &vars);
    float dist = rssDistance(&loc_R, &loc_t, &r1[t_j], &d2[t_j], &vars[t_j]);
    arr_res[g_j].dist = dist;
  }

}

int main(int argc, char *argv[])
{
  
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  cout << "Device name: " << prop.name << endl;
  cout << "float size is " << sizeof(float) << endl;

  test_rectangles_2D();
  test_rectangles_3D();
  test_stress_random_RSS();

  print_Stats();

  testSingleTiming();
  testMultipleSerial();
  testMultipleSame();
  testMultipleRandom();
}

HOST_PREFIX float distTrianglesGPU(const Triangle* h_s1, const Triangle* h_s2, DistTriangleVars*)
{
  return 0;
}

// use this function to verify working by running it against the test cases
HOST_PREFIX float distRSSGPU(const Matrix3* R, const Vector3* t,
            const RSS* h_r1, const RSS* h_r2, DistRSSVars* d)
{
  RSS *d_r1, *d_r2;
  Matrix3 *d_R;
  Vector3 *d_t;

  hipMalloc(&d_R, sizeof(Matrix3));
  hipMalloc(&d_t, sizeof(Vector3));
  hipMalloc(&d_r1, size_rss);
  hipMalloc(&d_r2, size_rss);
  Result *d_res;
  Result h_res;
  hipMalloc(&d_res, sizeof(Result));

  hipMemcpy(d_R, R, sizeof(Matrix3), hipMemcpyHostToDevice);
  hipMemcpy(d_t, t, sizeof(Vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_r1, h_r1, size_rss, hipMemcpyHostToDevice);
  hipMemcpy(d_r2, h_r2, size_rss, hipMemcpyHostToDevice);

  computeDistance<<<1, 1>>>(d_R, d_t, d_r1, d_r2, d_res);

  hipMemcpy(&h_res, d_res, sizeof(Result), hipMemcpyDeviceToHost);

  hipFree(d_r1);
  hipFree(d_r2);
  hipFree(d_res);

  return h_res.dist;

}

void testSingleTiming()
{
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  RSS h_r1, h_r2;
  RSS *d_r1, *d_r2;

  //TODO change this to be more general

  h_r1 = getRandomRSS();
  h_r2 = getRandomRSS();

  Matrix3 *d_R;
  Vector3 *d_t;

  hipMalloc(&d_R, sizeof(Matrix3));
  hipMalloc(&d_t, sizeof(Vector3));
  hipMalloc(&d_r1, size_rss);
  hipMalloc(&d_r2, size_rss);
  Result *d_res;
  Result h_res;
  hipMalloc(&d_res, sizeof(Result));
  
  double t_init = get_wall_time();

  hipMemcpy(d_R, &MAT_I, sizeof(Matrix3), hipMemcpyHostToDevice);
  hipMemcpy(d_t, &t0, sizeof(Vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_r1, &h_r1, size_rss, hipMemcpyHostToDevice);
  hipMemcpy(d_r2, &h_r2, size_rss, hipMemcpyHostToDevice);

  computeDistance<<<1, 1>>>(d_R, d_t, d_r1, d_r2, d_res);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(&h_res, d_res, sizeof(Result), hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);

  cout << setprecision(5);
  // cout << "Time for results using CUDA Event: " << t_results << "ms" <<  endl;
  cout << "Wall time (Single): " << (t_cuda_end - t_init )*1000 << "ms" <<  endl;
  cout << endl;

  hipFree(d_r1);
  hipFree(d_r2);
  hipFree(d_res);

}

void testMultipleSerial()
{
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  RSS h_r1, h_r2;
  RSS *d_r1, *d_r2;

  //TODO change this to be more general

  h_r1 = getRandomRSS();
  h_r2 = getRandomRSS();

  Matrix3 *d_R;
  Vector3 *d_t;

  hipMalloc(&d_R, sizeof(Matrix3));
  hipMalloc(&d_t, sizeof(Vector3));
  hipMalloc(&d_r1, size_rss);
  hipMalloc(&d_r2, size_rss);
  Result *d_res;
  Result h_res;
  hipMalloc(&d_res, sizeof(Result));
  
  double t_init = get_wall_time();

  hipMemcpy(d_R, &MAT_I, sizeof(Matrix3), hipMemcpyHostToDevice);
  hipMemcpy(d_t, &t0, sizeof(Vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_r1, &h_r1, size_rss, hipMemcpyHostToDevice);
  hipMemcpy(d_r2, &h_r2, size_rss, hipMemcpyHostToDevice);

  for(int i = 0; i < 1000; i++)
    computeDistance<<<1, 1>>>(d_R, d_t, d_r1, d_r2, d_res);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(&h_res, d_res, sizeof(Result), hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);

  cout << setprecision(5);
  // cout << "Time for results using CUDA Event: " << t_results << "ms" <<  endl;
  cout << "Wall time multiple serial (" << 1000 << "): " << (t_cuda_end - t_init )*1000 << "ms" <<  endl;
  cout << endl;

  hipFree(d_r1);
  hipFree(d_r2);
  hipFree(d_res);

}

void testMultipleSame()
{
  srand(static_cast<unsigned> (time(NULL)));
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  RSS *h_r1, *h_r2;
  RSS* d_r1, *d_r2;

  h_r1 = new RSS[NUM_CHECK];
  h_r2 = new RSS[NUM_CHECK];

  if(!h_r1 || !h_r2)
  {
    cout << "Could not initialize h_r, out of memory" << endl;
    return;
  }

  h_r1[0] = getRandomRSS();
  h_r2[0] = getRandomRSS();

  for(int i = 1; i < NUM_CHECK-1; i++)
  {
    h_r1[i] = h_r1[0];
    h_r2[i] = h_r2[0];
  }

  float actual_res = distRectangles_fcl(h_r1[0], h_r2[0]);
  // cout << "s1 " << h_r1[0];
  // cout << "s2 " << h_r2[0];
  // cout << "dist is " << actual_res << endl;

  // cout << "L: s1 " << h_r1[NUM_CHECK-1];
  // cout << "L: s2 " << h_r2[NUM_CHECK-1];
  // cout << "L: dist is " << distRSSs_fcl(h_r1[NUM_CHECK-1],
  //                            h_r2[NUM_CHECK-1]) << endl;
  Matrix3 *d_R;
  Vector3 *d_t;

  hipMalloc(&d_R, sizeof(Matrix3));
  hipMalloc(&d_t, sizeof(Vector3));
  hipMalloc(&d_r1, NUM_CHECK*size_rss);
  hipMalloc(&d_r2, NUM_CHECK*size_rss);
  Result *d_res;
  Result *h_res;
  h_res = new Result[NUM_CHECK];
  hipMalloc(&d_res, NUM_CHECK*sizeof(Result));
  
  double t_init = get_wall_time();

  hipMemcpy(d_R, &MAT_I, sizeof(Matrix3), hipMemcpyHostToDevice);
  hipMemcpy(d_t, &t0, sizeof(Vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_r1, h_r1, NUM_CHECK*size_rss, hipMemcpyHostToDevice);
  hipMemcpy(d_r2, h_r2, NUM_CHECK*size_rss, hipMemcpyHostToDevice);

  int numBlocks = (NUM_CHECK - 1)/BLOCKSIZE + 1;
  cout << "numBlocks is " << numBlocks << endl;
  dim3 dimBlock(1, BLOCKSIZE);
  dim3 dimGrid(1, numBlocks);
  computeDistanceArray<<<dimGrid, dimBlock>>>(d_R, d_t, d_r1, d_r2, d_res, NUM_CHECK);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(h_res, d_res, NUM_CHECK*sizeof(Result), hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);
  int count_correct = 0;
  for(int i = 0; i < NUM_CHECK; i++)
  {
    if(approx_equal(h_res[i].dist, actual_res))
      count_correct++;
    // cout << "actual: " << actual_res << " obtained: " << h_res[i].dist << endl;
  }

  cout << count_correct << "/" << NUM_CHECK << " are correct" << endl;

  cout << setprecision(5);
  cout << "Wall time multiple same(" << NUM_CHECK << "): " << (t_cuda_end - t_init )*1000 << "ms" <<  endl;
  cout << endl;

  hipFree(d_r1);
  hipFree(d_r2);
  hipFree(d_res);

  delete[] h_r1;
  delete[] h_r2;
  delete[] h_res;
}

void testMultipleRandom()
{
  srand(static_cast<unsigned> (time(NULL)));
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  RSS *h_r1, *h_r2;
  RSS* d_r1, *d_r2;

  h_r1 = new RSS[NUM_CHECK];
  h_r2 = new RSS[NUM_CHECK];

  if(!h_r1 || !h_r2)
  {
    cout << "Could not initialize h_r, out of memory" << endl;
    return;
  }

  cout << "Initializing the RSS ... " ;
  for(int i = 1; i < NUM_CHECK-1; i++)
  {
    h_r1[i] = getRandomRSS();
    h_r2[i] = getRandomRSS();
  }
  cout << "Done " << endl;
  // cout << "s1 " << h_r1[0];
  // cout << "s2 " << h_r2[0];
  // cout << "dist is " << actual_res << endl;

  // cout << "L: s1 " << h_r1[NUM_CHECK-1];
  // cout << "L: s2 " << h_r2[NUM_CHECK-1];
  // cout << "L: dist is " << distRSSs_fcl(h_r1[NUM_CHECK-1],
  //                            h_r2[NUM_CHECK-1]) << endl;
  Matrix3 *d_R;
  Vector3 *d_t;

  hipMalloc(&d_R, sizeof(Matrix3));
  hipMalloc(&d_t, sizeof(Vector3));
  hipMalloc(&d_r1, NUM_CHECK*size_rss);
  hipMalloc(&d_r2, NUM_CHECK*size_rss);
  Result *d_res;
  Result *h_res;
  h_res = new Result[NUM_CHECK];
  hipMalloc(&d_res, NUM_CHECK*sizeof(Result));
  
  double t_init = get_wall_time();

  hipMemcpy(d_R, &MAT_I, sizeof(Matrix3), hipMemcpyHostToDevice);
  hipMemcpy(d_t, &t0, sizeof(Vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_r1, h_r1, NUM_CHECK*size_rss, hipMemcpyHostToDevice);
  hipMemcpy(d_r2, h_r2, NUM_CHECK*size_rss, hipMemcpyHostToDevice);

  int numBlocks = (NUM_CHECK - 1)/BLOCKSIZE + 1;
  cout << "numBlocks is " << numBlocks << endl;
  dim3 dimBlock(1, BLOCKSIZE);
  dim3 dimGrid(1, numBlocks);
  computeDistanceArray<<<dimGrid, dimBlock>>>(d_R, d_t, d_r1, d_r2, d_res, NUM_CHECK);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(h_res, d_res, NUM_CHECK*sizeof(Result), hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);
  cout << "Evaluating reslts now ... " ;
  int count_correct = 0;
  for(int i = 0; i < NUM_CHECK; i++)
  {
    float actual_res = distRectangles_fcl(h_r1[i], h_r2[i]);

    if(approx_equal(h_res[i].dist, actual_res))
      count_correct++;
    // cout << "actual: " << actual_res << " obtained: " << h_res[i].dist << endl;
  }

  cout << " took " << (get_wall_time() - t_cuda_end)*1000 << "ms" << endl;

  cout << count_correct << "/" << NUM_CHECK << " are correct" << endl;
  cout << setprecision(5);
  cout << "Wall time multiple same(" << NUM_CHECK << "): " << (t_cuda_end - t_init )*1000 << "ms" <<  endl;
  cout << endl;

  hipFree(d_r1);
  hipFree(d_r2);
  hipFree(d_res);

  delete[] h_r1;
  delete[] h_r2;
  delete[] h_res;
}

HOST_PREFIX double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        //  Handle error
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}