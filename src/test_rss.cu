// @author : Atulya Shivam Shree
// @description : checks triangle intersection on CUDA

#include "compile_CUDA.h"

#include <cassert>
#include <iostream>
#include <iomanip>

#include "RSS.h"
float distTrianglesGPU(const Triangle*, const Triangle* , DistTriangleVars*);

float distRSSGPU(const Matrix3* R, const Vector3* t,
            const RSS* a, const RSS* b, DistRSSVars* d);

#define DIST_TRIANGLES distTrianglesGPU
// use the test cases on the GPU triangles function
#define DIST_RSS distRSSGPU
#include "Rectangle_tests.h"

#include "RSS-cuda-inl.h"

using namespace std;

const int size_rss = sizeof(RSS);

// single case timing
void testSingleTiming();
// multiple instances of same in sequence
void testMultipleSerial();
// multiple instances of same in an array
void testMultipleSame();
// Check collision within an array of random triangles
void testMultipleRandom();

int main(int argc, char *argv[])
{
  
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  cout << "Device name: " << prop.name << endl;
  cout << "float size is " << sizeof(float) << endl;

  test_rectangles_2D();
  test_rectangles_3D();
  test_stress_random_RSS();

  print_Stats();

  testSingleTiming();
  testMultipleSerial();
  testMultipleSame();
  testMultipleRandom();
}

HOST_PREFIX float distTrianglesGPU(const Triangle* h_s1, const Triangle* h_s2, DistTriangleVars*)
{
  return 0;
}

// use this function to verify working by running it against the test cases
HOST_PREFIX float distRSSGPU(const Matrix3* R, const Vector3* t,
            const RSS* h_r1, const RSS* h_r2, DistRSSVars* d)
{
  RSS *d_r1, *d_r2;
  Matrix3 *d_R;
  Vector3 *d_t;

  hipMalloc(&d_R, sizeof(Matrix3));
  hipMalloc(&d_t, sizeof(Vector3));
  hipMalloc(&d_r1, size_rss);
  hipMalloc(&d_r2, size_rss);
  RSSResult *d_res;
  RSSResult h_res;
  hipMalloc(&d_res, sizeof(RSSResult));

  hipMemcpy(d_R, R, sizeof(Matrix3), hipMemcpyHostToDevice);
  hipMemcpy(d_t, t, sizeof(Vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_r1, h_r1, size_rss, hipMemcpyHostToDevice);
  hipMemcpy(d_r2, h_r2, size_rss, hipMemcpyHostToDevice);

  computeDistance<<<1, 1>>>(d_R, d_t, d_r1, d_r2, d_res);

  hipMemcpy(&h_res, d_res, sizeof(RSSResult), hipMemcpyDeviceToHost);

  hipFree(d_r1);
  hipFree(d_r2);
  hipFree(d_res);

  return h_res.dist;

}

void testSingleTiming()
{
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  RSS h_r1, h_r2;
  RSS *d_r1, *d_r2;

  //TODO change this to be more general

  h_r1 = getRandomRSS();
  h_r2 = getRandomRSS();

  Matrix3 *d_R;
  Vector3 *d_t;

  hipMalloc(&d_R, sizeof(Matrix3));
  hipMalloc(&d_t, sizeof(Vector3));
  hipMalloc(&d_r1, size_rss);
  hipMalloc(&d_r2, size_rss);
  RSSResult *d_res;
  RSSResult h_res;
  hipMalloc(&d_res, sizeof(RSSResult));
  
  double t_init = get_wall_time();

  hipMemcpy(d_R, &MAT_I, sizeof(Matrix3), hipMemcpyHostToDevice);
  hipMemcpy(d_t, &t0, sizeof(Vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_r1, &h_r1, size_rss, hipMemcpyHostToDevice);
  hipMemcpy(d_r2, &h_r2, size_rss, hipMemcpyHostToDevice);

  computeDistance<<<1, 1>>>(d_R, d_t, d_r1, d_r2, d_res);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(&h_res, d_res, sizeof(RSSResult), hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);

  cout << setprecision(5);
  // cout << "Time for results using CUDA Event: " << t_results << "ms" <<  endl;
  cout << "Wall time (Single): " << (t_cuda_end - t_init )*1000 << "ms" <<  endl;
  cout << endl;

  hipFree(d_r1);
  hipFree(d_r2);
  hipFree(d_res);

}

void testMultipleSerial()
{
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  RSS h_r1, h_r2;
  RSS *d_r1, *d_r2;

  //TODO change this to be more general

  h_r1 = getRandomRSS();
  h_r2 = getRandomRSS();

  Matrix3 *d_R;
  Vector3 *d_t;

  hipMalloc(&d_R, sizeof(Matrix3));
  hipMalloc(&d_t, sizeof(Vector3));
  hipMalloc(&d_r1, size_rss);
  hipMalloc(&d_r2, size_rss);
  RSSResult *d_res;
  RSSResult h_res;
  hipMalloc(&d_res, sizeof(RSSResult));
  
  double t_init = get_wall_time();

  hipMemcpy(d_R, &MAT_I, sizeof(Matrix3), hipMemcpyHostToDevice);
  hipMemcpy(d_t, &t0, sizeof(Vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_r1, &h_r1, size_rss, hipMemcpyHostToDevice);
  hipMemcpy(d_r2, &h_r2, size_rss, hipMemcpyHostToDevice);

  for(int i = 0; i < 1000; i++)
    computeDistance<<<1, 1>>>(d_R, d_t, d_r1, d_r2, d_res);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(&h_res, d_res, sizeof(RSSResult), hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);

  cout << setprecision(5);
  // cout << "Time for results using CUDA Event: " << t_results << "ms" <<  endl;
  cout << "Wall time multiple serial (" << 1000 << "): " << (t_cuda_end - t_init )*1000 << "ms" <<  endl;
  cout << endl;

  hipFree(d_r1);
  hipFree(d_r2);
  hipFree(d_res);

}

void testMultipleSame()
{
  srand(static_cast<unsigned> (time(NULL)));
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  RSS *h_r1, *h_r2;
  RSS* d_r1, *d_r2;

  h_r1 = new RSS[NUM_CHECK];
  h_r2 = new RSS[NUM_CHECK];

  if(!h_r1 || !h_r2)
  {
    cout << "Could not initialize h_r, out of memory" << endl;
    return;
  }

  h_r1[0] = getRandomRSS();
  h_r2[0] = getRandomRSS();

  for(int i = 1; i < NUM_CHECK-1; i++)
  {
    h_r1[i] = h_r1[0];
    h_r2[i] = h_r2[0];
  }

  float actual_res = distRectangles_fcl(h_r1[0], h_r2[0]);
  // cout << "s1 " << h_r1[0];
  // cout << "s2 " << h_r2[0];
  // cout << "dist is " << actual_res << endl;

  // cout << "L: s1 " << h_r1[NUM_CHECK-1];
  // cout << "L: s2 " << h_r2[NUM_CHECK-1];
  // cout << "L: dist is " << distRSSs_fcl(h_r1[NUM_CHECK-1],
  //                            h_r2[NUM_CHECK-1]) << endl;
  Matrix3 *d_R;
  Vector3 *d_t;

  hipMalloc(&d_R, sizeof(Matrix3));
  hipMalloc(&d_t, sizeof(Vector3));
  hipMalloc(&d_r1, NUM_CHECK*size_rss);
  hipMalloc(&d_r2, NUM_CHECK*size_rss);
  RSSResult *d_res;
  RSSResult *h_res;
  h_res = new RSSResult[NUM_CHECK];
  hipMalloc(&d_res, NUM_CHECK*sizeof(RSSResult));
  
  double t_init = get_wall_time();

  hipMemcpy(d_R, &MAT_I, sizeof(Matrix3), hipMemcpyHostToDevice);
  hipMemcpy(d_t, &t0, sizeof(Vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_r1, h_r1, NUM_CHECK*size_rss, hipMemcpyHostToDevice);
  hipMemcpy(d_r2, h_r2, NUM_CHECK*size_rss, hipMemcpyHostToDevice);

  int numBlocks = (NUM_CHECK - 1)/BLOCKSIZE + 1;
  cout << "numBlocks is " << numBlocks << endl;
  dim3 dimBlock(1, BLOCKSIZE);
  dim3 dimGrid(1, numBlocks);
  computeDistanceArray<<<dimGrid, dimBlock>>>(d_R, d_t, d_r1, d_r2, d_res, NUM_CHECK);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(h_res, d_res, NUM_CHECK*sizeof(RSSResult), hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);
  int count_correct = 0;
  for(int i = 0; i < NUM_CHECK; i++)
  {
    if(approx_equal(h_res[i].dist, actual_res))
      count_correct++;
    // cout << "actual: " << actual_res << " obtained: " << h_res[i].dist << endl;
  }

  cout << count_correct << "/" << NUM_CHECK << " are correct" << endl;

  cout << setprecision(5);
  cout << "Wall time multiple same(" << NUM_CHECK << "): " << (t_cuda_end - t_init )*1000 << "ms" <<  endl;
  cout << endl;

  hipFree(d_r1);
  hipFree(d_r2);
  hipFree(d_res);

  delete[] h_r1;
  delete[] h_r2;
  delete[] h_res;
}

void testMultipleRandom()
{
  srand(static_cast<unsigned> (time(NULL)));
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  RSS *h_r1, *h_r2;
  RSS* d_r1, *d_r2;

  h_r1 = new RSS[NUM_CHECK];
  h_r2 = new RSS[NUM_CHECK];

  if(!h_r1 || !h_r2)
  {
    cout << "Could not initialize h_r, out of memory" << endl;
    return;
  }

  cout << "Initializing the RSS ... " ;
  for(int i = 1; i < NUM_CHECK-1; i++)
  {
    h_r1[i] = getRandomRSS();
    h_r2[i] = getRandomRSS();
  }
  cout << "Done " << endl;
  // cout << "s1 " << h_r1[0];
  // cout << "s2 " << h_r2[0];
  // cout << "dist is " << actual_res << endl;

  // cout << "L: s1 " << h_r1[NUM_CHECK-1];
  // cout << "L: s2 " << h_r2[NUM_CHECK-1];
  // cout << "L: dist is " << distRSSs_fcl(h_r1[NUM_CHECK-1],
  //                            h_r2[NUM_CHECK-1]) << endl;
  Matrix3 *d_R;
  Vector3 *d_t;

  hipMalloc(&d_R, sizeof(Matrix3));
  hipMalloc(&d_t, sizeof(Vector3));
  hipMalloc(&d_r1, NUM_CHECK*size_rss);
  hipMalloc(&d_r2, NUM_CHECK*size_rss);
  RSSResult *d_res;
  RSSResult *h_res;
  h_res = new RSSResult[NUM_CHECK];
  hipMalloc(&d_res, NUM_CHECK*sizeof(RSSResult));
  
  double t_init = get_wall_time();

  hipMemcpy(d_R, &MAT_I, sizeof(Matrix3), hipMemcpyHostToDevice);
  hipMemcpy(d_t, &t0, sizeof(Vector3), hipMemcpyHostToDevice);
  hipMemcpy(d_r1, h_r1, NUM_CHECK*size_rss, hipMemcpyHostToDevice);
  hipMemcpy(d_r2, h_r2, NUM_CHECK*size_rss, hipMemcpyHostToDevice);

  int numBlocks = (NUM_CHECK - 1)/BLOCKSIZE + 1;
  cout << "numBlocks is " << numBlocks << endl;
  dim3 dimBlock(1, BLOCKSIZE);
  dim3 dimGrid(1, numBlocks);
  computeDistanceArray<<<dimGrid, dimBlock>>>(d_R, d_t, d_r1, d_r2, d_res, NUM_CHECK);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(h_res, d_res, NUM_CHECK*sizeof(RSSResult), hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);
  cout << "Evaluating reslts now ... " ;
  int count_correct = 0;
  for(int i = 0; i < NUM_CHECK; i++)
  {
    float actual_res = distRectangles_fcl(h_r1[i], h_r2[i]);

    if(approx_equal(h_res[i].dist, actual_res))
      count_correct++;
    // cout << "actual: " << actual_res << " obtained: " << h_res[i].dist << endl;
  }

  cout << " took " << (get_wall_time() - t_cuda_end)*1000 << "ms" << endl;

  cout << count_correct << "/" << NUM_CHECK << " are correct" << endl;
  cout << setprecision(5);
  cout << "Wall time multiple random(" << NUM_CHECK << "): " << (t_cuda_end - t_init )*1000 << "ms" <<  endl;
  cout << endl;

  hipFree(d_r1);
  hipFree(d_r2);
  hipFree(d_res);

  delete[] h_r1;
  delete[] h_r2;
  delete[] h_res;
}