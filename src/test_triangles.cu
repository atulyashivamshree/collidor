// @author : Atulya Shivam Shree
// @description : checks triangle intersection on CUDA

#include "compile_CUDA.h"

#include <cassert>
#include <iostream>
#include <iomanip>

#define BLOCKSIZE 32

#include "Triangle.h"
float distTrianglesGPU(const Triangle*, const Triangle* ,
                       const float R[3][3], const float t[3], 
                       DistTriangleVars*);

// use the test cases on the GPU triangles function
#define DIST_TRIANGLES distTrianglesGPU
#include "Triangles_test.h"

#include "Triangle-cuda-inl.h"

using namespace std;

const int size_tri = sizeof(Triangle);

// single case timing
void testSingleTiming();
// multiple instances of same in sequence
void testMultipleSerial();
// multiple instances of same in an array
void testMultipleSame();
// Check collision within an array of random triangles
void testMultipleRandom();

__host__ int main(int argc, char *argv[])
{
  
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);
  cout << "Device name: " << prop.name << endl;
  cout << "float size is " << sizeof(float) << endl;

  test_triangles_2D();
  test_triangles_3D();
  // test_stress_random();

  testSingleTiming();
  testMultipleSerial();
  testMultipleSame();
  testMultipleRandom();
}

HOST_PREFIX float distTrianglesGPU(const Triangle* h_s1, const Triangle* h_s2,
                            const float R[3][3], const float t[3], 
                            DistTriangleVars*)
{
  Triangle *d_s1, *d_s2;

  hipMalloc(&d_s1, size_tri);
  hipMalloc(&d_s2, size_tri);
  TriangleResult *d_res;
  TriangleResult h_res;
  hipMalloc(&d_res, sizeof(TriangleResult));

  hipMemcpy(d_s1, h_s1, size_tri, hipMemcpyHostToDevice);
  hipMemcpy(d_s2, h_s2, size_tri, hipMemcpyHostToDevice);

  computeDistanceSingle<<<1, 1>>>(d_s1, d_s2, d_res);

  hipMemcpy(&h_res, d_res, sizeof(TriangleResult), hipMemcpyDeviceToHost);

  hipFree(d_s1);
  hipFree(d_s2);
  hipFree(d_res);

  return h_res.dist;

}

void testSingleTiming()
{
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  Triangle h_s1, h_s2;
  Triangle *d_s1, *d_s2;

  generateRandomTriangle(&h_s1);
  generateRandomTriangle(&h_s2);

  hipMalloc(&d_s1, size_tri);
  hipMalloc(&d_s2, size_tri);
  TriangleResult *d_res;
  TriangleResult h_res;
  hipMalloc(&d_res, sizeof(TriangleResult));
  
  double t_init = get_wall_time();

  hipMemcpy(d_s1, &h_s1, size_tri, hipMemcpyHostToDevice);
  hipMemcpy(d_s2, &h_s2, size_tri, hipMemcpyHostToDevice);

  computeDistanceSingle<<<1, 1>>>(d_s1, d_s2, d_res);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(&h_res, d_res, sizeof(TriangleResult), hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);

  cout << setprecision(5);
  // cout << "Time for results using CUDA Event: " << t_results << "ms" <<  endl;
  cout << "Wall time (Single): " << (t_cuda_end - t_init )*1000 << "ms" <<  endl;
  cout << endl;

  hipFree(d_s1);
  hipFree(d_s2);
  hipFree(d_res);

}

void testMultipleSerial()
{
  srand(static_cast<unsigned> (time(NULL)));
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  Triangle h_s1, h_s2;
  Triangle *d_s1, *d_s2;

  generateRandomTriangle(&h_s1);
  generateRandomTriangle(&h_s2);

  hipMalloc(&d_s1, size_tri);
  hipMalloc(&d_s2, size_tri);
  TriangleResult *d_res;
  TriangleResult h_res;
  hipMalloc(&d_res, sizeof(TriangleResult));
  
  double t_init = get_wall_time();

  hipMemcpy(d_s1, &h_s1, size_tri, hipMemcpyHostToDevice);
  hipMemcpy(d_s2, &h_s2, size_tri, hipMemcpyHostToDevice);

  for(int i = 0; i < NUM_CHECK; i++)
    computeDistanceSingle<<<1, 1>>>(d_s1, d_s2, d_res);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(&h_res, d_res, sizeof(TriangleResult), hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);

  cout << setprecision(5);
  cout << "Wall time multiple serial (" << NUM_CHECK << "): " << (t_cuda_end - t_init )*1000 << "ms" <<  endl;
  cout << endl;

  hipFree(d_s1);
  hipFree(d_s2);
  hipFree(d_res);

}

void testMultipleSame()
{
  srand(static_cast<unsigned> (time(NULL)));
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  Triangle *h_s1, *h_s2;
  Triangle* d_s1, *d_s2;

  h_s1 = new Triangle[NUM_CHECK];
  h_s2 = new Triangle[NUM_CHECK];

  generateRandomTriangle(&h_s1[0]);
  generateRandomTriangle(&h_s2[0]);

  for(int i = 1; i < NUM_CHECK-1; i++)
  {
    h_s1[i] = h_s1[0];
    h_s2[i] = h_s2[0];
  }

  float actual_res = distTriangles_fcl(h_s1[0], h_s2[0], matI_, t0_);
  // cout << "s1 " << h_s1[0];
  // cout << "s2 " << h_s2[0];
  // cout << "dist is " << actual_res << endl;

  // cout << "L: s1 " << h_s1[NUM_CHECK-1];
  // cout << "L: s2 " << h_s2[NUM_CHECK-1];
  // cout << "L: dist is " << distTriangles_fcl(h_s1[NUM_CHECK-1, matI_, t0_],
  //                            h_s2[NUM_CHECK-1]) << endl;

  hipMalloc(&d_s1, NUM_CHECK*size_tri);
  hipMalloc(&d_s2, NUM_CHECK*size_tri);
  TriangleResult *d_res;
  TriangleResult *h_res = new TriangleResult[NUM_CHECK];
  hipMalloc(&d_res, NUM_CHECK*sizeof(TriangleResult));
  
  double t_init = get_wall_time();

  hipMemcpy(d_s1, h_s1, NUM_CHECK*size_tri, hipMemcpyHostToDevice);
  hipMemcpy(d_s2, h_s2, NUM_CHECK*size_tri, hipMemcpyHostToDevice);

  int numBlocks = (NUM_CHECK - 1)/BLOCKSIZE + 1;
  dim3 dimBlock(1, BLOCKSIZE);
  dim3 dimGrid(1, numBlocks);
  computeDistanceArray<<<dimGrid, dimBlock>>>(d_s1, d_s2, d_res, NUM_CHECK);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(h_res, d_res, NUM_CHECK*sizeof(TriangleResult), hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);
  int count_correct = 0;
  for(int i = 0; i < NUM_CHECK; i++)
  {
    if(approx_equal(h_res[i].dist, actual_res))
      count_correct++;
    // cout << "actual: " << actual_res << " obtained: " << h_res[i].dist << endl;
  }

  cout << count_correct << "/" << NUM_CHECK << " are correct" << endl;

  cout << setprecision(5);
  cout << "Wall time multiple same(" << NUM_CHECK << "): " << (t_cuda_end - t_init )*1000 << "ms" <<  endl;
  cout << endl;

  hipFree(d_s1);
  hipFree(d_s2);
  hipFree(d_res);

  delete[] h_s1;
  delete[] h_s2;
  delete[] h_res;

}

void testMultipleRandom()
{
  srand(static_cast<unsigned> (time(NULL)));
  // initialize the dimension blocks for the differnet kernels
  double t_start = get_wall_time();

  // A is the input to every stage while C is the output after every stage
  Triangle *h_s1, *h_s2;
  Triangle* d_s1, *d_s2;

  h_s1 = new Triangle[NUM_CHECK];
  h_s2 = new Triangle[NUM_CHECK];

  for(int i = 0; i < NUM_CHECK; i++)
  {
    generateRandomTriangle(&h_s1[i]);
    generateRandomTriangle(&h_s2[i]);
  }

  float actual_res = distTriangles_fcl(h_s1[0], h_s2[0], matI_, t0_);

  hipMalloc(&d_s1, NUM_CHECK*size_tri);
  hipMalloc(&d_s2, NUM_CHECK*size_tri);
  TriangleResult *d_res;
  TriangleResult *h_res = new TriangleResult[NUM_CHECK];
  hipMalloc(&d_res, NUM_CHECK*sizeof(TriangleResult));
  
  double t_init = get_wall_time();

  hipMemcpy(d_s1, h_s1, NUM_CHECK*size_tri, hipMemcpyHostToDevice);
  hipMemcpy(d_s2, h_s2, NUM_CHECK*size_tri, hipMemcpyHostToDevice);

  int numBlocks = (NUM_CHECK - 1)/BLOCKSIZE + 1;
  dim3 dimBlock(1, BLOCKSIZE);
  dim3 dimGrid(1, numBlocks);
  computeDistanceArray<<<dimGrid, dimBlock>>>(d_s1, d_s2, d_res, NUM_CHECK);

  hipDeviceSynchronize();
  double t_cuda_end = get_wall_time();

  hipMemcpy(h_res, d_res, NUM_CHECK*sizeof(TriangleResult), hipMemcpyDeviceToHost);

  // float t_results;
  // hipEventElapsedTime(&t_results, start, results);
  int count_correct = 0;
  for(int i = 0; i < NUM_CHECK; i++)
  {
    float actual_dist = distTriangles_fcl(h_s1[i], h_s2[i], matI_, t0_);
    if(approx_equal(h_res[i].dist, actual_dist))
      count_correct++;
    // cout << "actual: " << actual_res << " obtained: " << h_res[i].dist << endl;
  }

  cout << count_correct << "/" << NUM_CHECK << " are correct" << endl;

  cout << setprecision(5);
  cout << "Wall time multiple random(" << NUM_CHECK << "): " << (t_cuda_end - t_init )*1000 << "ms" <<  endl;
  cout << endl;

  hipFree(d_s1);
  hipFree(d_s2);
  hipFree(d_res);

  delete[] h_s1;
  delete[] h_s2;
  delete[] h_res;

}